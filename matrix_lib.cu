#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

extern int threadsPerBlock;
extern int blocksPerGrid;

__global__ void scalar_matrix_mult_kernel(float scalar_value, float *matrix_rows, unsigned long int num_elements)
{
    unsigned long int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_elements)
    {
        matrix_rows[idx] *= scalar_value;
    }
}

__global__ void matrix_matrix_mult_kernel(float *matrixA, float *matrixB, float *matrixC, unsigned long int heightA, unsigned long int widthA, unsigned long int widthB)
{
    unsigned long int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < heightA && col < widthB)
    {
        float value = 0;
        for (unsigned long int k = 0; k < widthA; k++)
        {
            value += matrixA[row * widthA + k] * matrixB[k * widthB + col];
        }
        matrixC[row * widthB + col] = value;
    }
}

int scalar_matrix_mult(float scalar_value, struct matrix *matrix)
{
    hipError_t hipError_t;

    // Verificar se a matriz passada é válida
    if (matrix == NULL || matrix->rows == NULL)
    {
        printf("Erro: Matriz inválida.\n");
        return 0;
    }

    // Determinar o número de elementos na matriz
    unsigned long int num_elements = matrix->height * matrix->width;

    // Alocar memória na GPU para a matriz
    float *d_matrix_rows;
    hipError_t = hipMalloc(&d_matrix_rows, num_elements * sizeof(float));
    if (hipError_t != hipSuccess)
    {
        printf("hipMalloc d_x returned error %s (code %d)\n",
               hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    // Copiar os dados da matriz da CPU para a GPU
    hipError_t = hipMemcpy(d_matrix_rows, matrix->rows, num_elements * sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (h_x -> d_x) returned error %s (code %d), line(% d)\n ", hipGetErrorString(hipError_t), hipError_t,
               __LINE__);
        return 0;
    }

    // Lançar o kernel
    scalar_matrix_mult_kernel<<<blocksPerGrid, threadsPerBlock>>>(scalar_value, d_matrix_rows, num_elements);

    // Esperar a execução do kernel terminar
    hipDeviceSynchronize();

    hipGetLastError();

    // Copiar os resultados da GPU de volta para a CPU
    hipError_t = hipMemcpy(matrix->rows, d_matrix_rows, num_elements * sizeof(float), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (d_x -> h_y) returned error %s (code %d), line(% d)\n ", hipGetErrorString(hipError_t),
               hipError_t,
               __LINE__);
        return 0;
    }

    // Liberar a memória na GPU
    hipFree(d_matrix_rows);

    // Retornar 1 indicando que a operação foi bem-sucedida
    return 1;
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix *matrixB, struct matrix *matrixC)
{
    hipError_t hipError_t;

    // Verificar se as dimensões são válidas para a multiplicação de matrizes
    if (matrixA->width != matrixB->height)
    {
        printf("Erro: As dimensões das matrizes não são compatíveis para multiplicação.\n");
        return 0;
    }

    unsigned long int heightA = matrixA->height;
    unsigned long int widthA = matrixA->width;
    unsigned long int widthB = matrixB->width;

    // Alocar memória na GPU para as matrizes
    float *d_matrixA, *d_matrixB, *d_matrixC;
    hipError_t = hipMalloc(&d_matrixA, heightA * widthA * sizeof(float));
    if (hipError_t != hipSuccess)
    {
        printf("hipMalloc d_x returned error %s (code %d)\n",
               hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    hipError_t = hipMalloc(&d_matrixB, widthA * widthB * sizeof(float));
    if (hipError_t != hipSuccess)
    {
        printf("hipMalloc d_x returned error %s (code %d)\n",
               hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    hipError_t = hipMalloc(&d_matrixC, heightA * widthB * sizeof(float));
    if (hipError_t != hipSuccess)
    {
        printf("hipMalloc d_x returned error %s (code %d)\n",
               hipGetErrorString(hipError_t), hipError_t);
        return 0;
    }

    // Copiar os dados das matrizes da CPU para a GPU
    hipError_t = hipMemcpy(d_matrixA, matrixA->rows, heightA * widthA * sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (h_x -> d_x) returned error %s (code %d), line(% d)\n ", hipGetErrorString(hipError_t), hipError_t,
               __LINE__);
        return 0;
    }

    hipError_t = hipMemcpy(d_matrixB, matrixB->rows, widthA * widthB * sizeof(float), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (h_x -> d_x) returned error %s (code %d), line(% d)\n ", hipGetErrorString(hipError_t), hipError_t,
               __LINE__);
        return 0;
    }

    // Definir o número de threads por bloco e o número de blocos
    // dim3 threadsPerBlock(16, 16);
    // dim3 blocksPerGrid((widthB + threadsPerBlock.x - 1) / threadsPerBlock.x, (heightA + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Lançar o kernel
    matrix_matrix_mult_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_matrixA, d_matrixB, d_matrixC, heightA, widthA, widthB);

    // Esperar a execução do kernel terminar
    hipDeviceSynchronize();

    hipGetLastError();

    // Copiar os resultados da GPU de volta para a CPU
    hipError_t = hipMemcpy(matrixC->rows, d_matrixC, heightA * widthB * sizeof(float), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess)
    {
        printf("hipMemcpy (h_x -> d_x) returned error %s (code %d), line(% d)\n ", hipGetErrorString(hipError_t), hipError_t,
               __LINE__);
        return 0;
    }

    // Liberar a memória na GPU
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);

    return 1;
}